// sample/vector_add.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *a, const float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) c[idx] = a[idx] + b[idx];
}

int main() {
    const int n = 256;
    size_t size = n * sizeof(float);

    float *h_a = (float*)malloc(size);
    float *h_b = (float*)malloc(size);
    float *h_c = (float*)malloc(size);

    for (int i = 0; i < n; ++i) {
        h_a[i] = float(i);
        h_b[i] = float(i * 2);
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threads = 128;
    int blocks = (n + threads - 1) / threads;
    vectorAdd<<<blocks, threads>>>(d_a, d_b, d_c, n);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) {
        printf("%f + %f = %f\n", h_a[i], h_b[i], h_c[i]);
    }

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(h_a); free(h_b); free(h_c);
    return 0;
}
